
#include <hip/hip_runtime.h>
#include <stdlib.h> 
#include <string.h> 
#include <time.h>
#include <stdio.h>
__global__ void sumArray(float *A, float *B, float *C) { 
	int i = blockIdx.x;
	C[i] = A[i] + B[i];
}

void initialData(float *ip, int size){
	time_t t;
	srand((unsigned int)time(&t));
	for(int i = 0; i < size; i++){
		ip[i] = (float)( rand() & 0xFF )/10.0f;
	}
}
int main(void){
	int nElem = 16;
	size_t nBytes = nElem * sizeof(float);
	float *h_A, *h_B, *h_C;
	float *d_A, *d_B, *d_C;
	h_A = (float *)malloc(nBytes); h_B = (float *)malloc(nBytes); h_C = (float *)malloc(nBytes);
	hipMalloc((float**)&d_A, nBytes);
	hipMalloc((float**)&d_B, nBytes);
	hipMalloc((float**)&d_C, nBytes);
	initialData(h_A, nElem); initialData(h_B, nElem);
	hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice);
	dim3 block (nElem);
	dim3 grid (nElem/block.x);
	sumArray<<<block,grid>>>(d_A,d_B,d_C);
	printf("\nExecution configuration <<<%d, %d>>>\n",grid.x,block.x);
	hipMemcpy(h_C, d_C, nBytes, hipMemcpyDeviceToHost);
	for(int i = 0; i < nElem; i ++){
		printf("|%f",h_A[i]);
	}
	printf("\n");
	for(int i = 0; i < nElem; i++){
		printf("|%f",h_B[i]);
	}
	printf("\n");
	for(int i = 0; i < nElem; i++){
		printf("|%f",h_C[i]);	
	}
	printf("\n");
	free(h_A); free(h_B); free(h_C);
	hipFree(d_A); hipFree(d_B); hipFree(d_C);
	return(0);
}
